
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

void chk_args(int argc, char **argv){
    if(argc <= 2){
        printf("Usage: %s filename out_name\n", argv[0]);
        exit(-1);
    }
}

/** 
 * Loads the color channel from file into the array.
 * Inputs:
     - FILE *img: The already opened image file.
     - float *arr: The color channel.
     - long M: The size of the row of the image.
     - long N: The size of the column of the image.
 * Output:
     - void.
**/
void load_row(FILE *img, float *arr, long M, long N){
    for(long y = 0; y < M*N; ++y){
        if(fscanf(img, "%f", &arr[y]) != 1){
            fprintf(stderr, "Error while reading\n");
            exit(-2);
        }
    }
}

/** 
 * Reads the file and stores its RGB values in arrays
 * Inputs:
     - char *filename: The name of the file to read.
     - float **r_arr: Here will be stored the array of the red channel. The image is linearized. Each image is next to each other.
     - float **g_arr: Here will be stored the array of the green channel. The image is linearized. Each image is next to each other.
     - float **b_arr: Here will be stored the array of the blue channel. The image is linearized. Each image is next to each other.
     - long *M: Here will be stored the amounts of rows per image.
     - long *N: Here will be stored the amounts of columns per image.
 * Output:
     - void.
**/
void load_file(char *filename, float **r_arr, float **g_arr, float **b_arr, long *M, long *N){
    FILE *img = fopen(filename, "r");
    fscanf(img, "%li %li", M, N);
    *r_arr = (float *)malloc(sizeof(float) * (*M)*(*N));
    *g_arr = (float *)malloc(sizeof(float) * (*M)*(*N));
    *b_arr = (float *)malloc(sizeof(float) * (*M)*(*N));

    load_row(img, *r_arr, *M, *N);
    load_row(img, *g_arr, *M, *N);
    load_row(img, *b_arr, *M, *N);

    fclose(img);
}

void write_file(char *outname, long M, long N, float *r_arr, float *g_arr, float *b_arr){
    FILE *out_file = fopen(outname, "w");
    fprintf(out_file, "%li %li\n", M, N);

    for(long i = 0; i < M*N-1; ++i){
        fprintf(out_file, "%f ", r_arr[i]);
    }
    fprintf(out_file, "%f\n", r_arr[M*N-1]);
    for(long i = 0; i < M*N-1; ++i){
        fprintf(out_file, "%f ", g_arr[i]);
    }
    fprintf(out_file, "%f\n", g_arr[M*N-1]);
    for(long i = 0; i < M*N-1; ++i){
        fprintf(out_file, "%f ", b_arr[i]);
    }
    fprintf(out_file, "%f\n", b_arr[M*N-1]);

    fclose(out_file);
}

__global__ void intercalar(float *src_arr, long M, long N, long x){
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    if(tId < N/x/2*M){
        for(long i = 0; i < x; ++i){
            float temp = src_arr[tId*(2*x) + i];
            src_arr[tId*(2*x) + i] = src_arr[tId*(2*x) + i + x];
            src_arr[tId*(2*x) + i + x] = temp;
        }
    }
}

void generar_imagen(char *out_name, float *r_arr_gpu, float *g_arr_gpu, float *b_arr_gpu, long M, long N, long x){
    long block_size = 256;
    long grid_size = (long)ceil((float)N/x/2*M/block_size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    intercalar<<<grid_size, block_size>>>(r_arr_gpu, M, N, x);
    intercalar<<<grid_size, block_size>>>(g_arr_gpu, M, N, x);
    intercalar<<<grid_size, block_size>>>(b_arr_gpu, M, N, x);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf ("%f[ms]\n", milliseconds); 

    float *new_r_arr = (float *)malloc(M*N * sizeof(float));
    float *new_g_arr = (float *)malloc(M*N * sizeof(float));
    float *new_b_arr = (float *)malloc(M*N * sizeof(float));
    hipMemcpy(new_r_arr, r_arr_gpu, sizeof(float)*M*N, hipMemcpyDeviceToHost);
    hipMemcpy(new_g_arr, g_arr_gpu, sizeof(float)*M*N, hipMemcpyDeviceToHost);
    hipMemcpy(new_b_arr, b_arr_gpu, sizeof(float)*M*N, hipMemcpyDeviceToHost);

    write_file(out_name, M, N, new_r_arr, new_g_arr, new_b_arr);

    free(new_r_arr);
    free(new_g_arr);
    free(new_b_arr);
}


int main(int argc, char **argv){
    chk_args(argc, argv);
    float *r_arr, *g_arr, *b_arr;
    long M, N;
    load_file(argv[1], &r_arr, &g_arr, &b_arr, &M, &N);
    
    /* CUDA SETUP */
    float *r_arr_gpu;
    float *g_arr_gpu;
    float *b_arr_gpu;
    hipMalloc(&r_arr_gpu, M*N * sizeof(float));
    hipMalloc(&g_arr_gpu, M*N * sizeof(float));
    hipMalloc(&b_arr_gpu, M*N * sizeof(float));
    
    hipMemcpy(r_arr_gpu, r_arr, M*N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(g_arr_gpu, g_arr, M*N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_arr_gpu, b_arr, M*N * sizeof(float), hipMemcpyHostToDevice);

    /* CUDA SETUP END */

    char *dst_name = (char *)malloc(strlen(argv[2]) + 3);
    strcpy(&dst_name[2], argv[2]);
    dst_name[1] = '_';

    for(long i = 0; i < 10; ++i){
        dst_name[0] = i + '0';
        long x = i == 0 ? 1 : 2 << (i-1);

        generar_imagen(dst_name, r_arr_gpu, g_arr_gpu, b_arr_gpu, M, N, x);
    }
    
    free(dst_name);

    hipFree(r_arr_gpu);
    hipFree(g_arr_gpu);
    hipFree(b_arr_gpu);

    return 0;
}
