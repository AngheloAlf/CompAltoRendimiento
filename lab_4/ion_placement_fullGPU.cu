#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define SIZE_MALLA 8192//1024
#define BLOCK_SIZE 256//1024
#define INITIAL_IONS 5000
#define MAX_IONS 6000
#define PBSTR "||||||||||||||||||||||||||||||||||||||||||||||||||||||||||||"
#define PBWIDTH 60

__constant__ float dev_ini_ions_xs[INITIAL_IONS];
__constant__ float dev_ini_ions_ys[INITIAL_IONS];

float uniform_rand(){
    return SIZE_MALLA*((float) rand() / (RAND_MAX));
}

void configSeed(unsigned seed){
    if(seed == 0){
        seed = (unsigned)time(NULL);
    }
    srand(seed);
    printf("results for seed %i\n", seed);  
}

void populate(float* ions_xs, float* ions_ys){
    for(int i = 0; i<INITIAL_IONS; i++){
        ions_xs[i]=uniform_rand();
        ions_ys[i]=uniform_rand();
    }
}

__device__ float distance(float p1x, float p1y, float p2x, float p2y){
    return sqrtf(powf(p1x-p2x, 2) + powf(p1y-p2y, 2));
}

__device__ void set_partials(float* mins, int* position, int tId){
    for(unsigned int s = blockDim.x/2; s>0; s>>=1){
        if(tId < s){
            if(mins[tId] > mins[tId+s]){
                mins[tId] = mins[tId+s];
                position[tId] = position[tId+s];
            }
        }
        __syncthreads();
    }
}

__global__ void set_Qs(float* dev_Q){
    int uThId = threadIdx.x + blockDim.x * blockIdx.x;
    float q = 0;
    for(int i = 0; i<INITIAL_IONS; i++){
        float dist = distance((float)(uThId%SIZE_MALLA), (float)(uThId/SIZE_MALLA), dev_ini_ions_xs[i], dev_ini_ions_ys[i]);
        if(dist == 0){
            q=INFINITY;
            break;
        }
        else{
            q+= 1/dist;
        }
    }
    dev_Q[uThId] = q;
}

__global__ void update_Qs_by_chunk(float* dev_Q, int i, int chunk_size){
    int uThId = threadIdx.x + blockDim.x * blockIdx.x;
    float q = 0;
    for(int n = i*chunk_size; n<(i+1)*chunk_size; n++){
        float dist = distance((float)(uThId%SIZE_MALLA), (float)(uThId/SIZE_MALLA), dev_ini_ions_xs[n], dev_ini_ions_ys[n]);
        if(dist == 0){
            q=INFINITY;
            break;
        }
        else{
            q += 1/dist;
        }
    }
    dev_Q[uThId]+= q;
}

__global__ void set_Qs_r(float* dev_Q, float r){
    int uThId = threadIdx.x + blockDim.x * blockIdx.x;
    float q = 0;
    for(int i = 0; i<INITIAL_IONS; i++){
        float dist = distance((float)(uThId%SIZE_MALLA), (float)(uThId/SIZE_MALLA), dev_ini_ions_xs[i], dev_ini_ions_ys[i]);
        if(dist<r){
            if(dist == 0){
                q=INFINITY;
                break;
            }
            else{
                q+= 1/dist;
            }
        }
    }
    dev_Q[uThId] = q;
}

__global__ void set_new_Ion(int new_ionIdx, float* dev_Q , float* new_ions_xs, float* new_ions_ys, float* partial_min, int* partial_min_pos){
    int tId = threadIdx.x;
    extern __shared__ float sdata[];
    float* mins = sdata;
    int* position = (int*)&sdata[blockDim.x];
    mins[tId] = partial_min[tId];
    position[tId] = partial_min_pos[tId];
    __syncthreads();
    set_partials(mins, position, tId);
    if(tId == 0){
        new_ions_xs[new_ionIdx] = (float)(position[0]%SIZE_MALLA);
        new_ions_ys[new_ionIdx] = (float)(position[0]/SIZE_MALLA);
        dev_Q[position[0]] = INFINITY;
    }
}

__global__ void partial_reduction(float* partial_min, int* partial_min_pos){
    int uThId = threadIdx.x + blockDim.x*blockIdx.x;
    int tId = threadIdx.x;
    extern __shared__ float sdata[];
    float* mins = sdata;
    int* position = (int*)&sdata[blockDim.x];
    mins[tId] = partial_min[uThId];
    position[tId] = partial_min_pos[uThId];
    __syncthreads();
    set_partials(mins, position, tId);
    if(tId == 0){
        partial_min[blockIdx.x] = mins[0];
        partial_min_pos[blockIdx.x] = position[0];
    }
}

__global__ void Q_reduction(float* dev_Q, float* partial_min, int* partial_min_pos){
    int uThId = threadIdx.x + blockDim.x*blockIdx.x;
    int tId = threadIdx.x;
    extern __shared__ float sdata[];
    float* mins = sdata;
    int* position = (int*)&sdata[blockDim.x];
    mins[tId] = dev_Q[uThId];
    position[tId] = uThId;
    __syncthreads();
    set_partials(mins, position, tId);
    if(tId == 0){
        partial_min[blockIdx.x] = mins[0];
        partial_min_pos[blockIdx.x] = position[0];
    }
}

__global__ void update_Qs(int new_ionIdx, float* dev_Q, float* new_ions_xs, float* new_ions_ys){
    int uThId = threadIdx.x + blockDim.x * blockIdx.x;
    float dist = distance((float)(uThId%SIZE_MALLA), (float)(uThId/SIZE_MALLA), new_ions_xs[new_ionIdx], new_ions_ys[new_ionIdx]);
    if(dist == 0){
        dev_Q[uThId]=INFINITY;
    }
    else{
        dev_Q[uThId] += 1/dist;
    }
}

__global__ void update_Qs_r(int new_ionIdx, float* dev_Q, float* new_ions_xs, float* new_ions_ys, float r){
    int uThId = threadIdx.x + blockDim.x * blockIdx.x;
    float dist = distance((float)(uThId%SIZE_MALLA), (float)(uThId/SIZE_MALLA), new_ions_xs[new_ionIdx], new_ions_ys[new_ionIdx]);
    if(dist<r){
        if(dist == 0){
            dev_Q[uThId]=INFINITY;
        }
        else{
            dev_Q[uThId] += 1/dist;
        }
    }
}

void setIon(int i, float* dev_Q, float* new_ions_xs, float* new_ions_ys, float* partial_min, int* partial_min_pos){
    int size;
    Q_reduction<<<SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE*(sizeof(float) + sizeof(int))>>>(dev_Q, partial_min, partial_min_pos);
    //printf("Ion %i\n", i);
    for(size = SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE; size>BLOCK_SIZE; size/=BLOCK_SIZE){
        //printf("size = %i\n", size);
        partial_reduction<<<size/BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE*(sizeof(float) + sizeof(int))>>>(partial_min, partial_min_pos);
        int inf = hipDeviceSynchronize();
        if(inf != 0){printf("fail in partial_reduction, cuda code: %i\n", inf);}
    }
    //printf("size = %i\n", size);
    int sMemSize = size*(sizeof(float) + sizeof(int));
    set_new_Ion<<<1, size, sMemSize>>>(i, dev_Q, new_ions_xs, new_ions_ys, partial_min, partial_min_pos);
    int inf = hipDeviceSynchronize();
    if(inf != 0){printf("fail in set_new_ion, cuda code: %i\n", inf);}
}

void printProgress (double percentage)
{
    int val = (int) (percentage * 100);
    int lpad = (int) (percentage * PBWIDTH);
    int rpad = PBWIDTH - lpad;
    printf ("\r%3d%% [%.*s%*s]", val, lpad, PBSTR, rpad, "");
    fflush (stdout);
}

int set_Qs_in_chunks(float* dev_Q){
    int chunk_size = 50;
    for(int i = 0; i<INITIAL_IONS/chunk_size; i++){
        update_Qs_by_chunk<<<SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE, BLOCK_SIZE>>>(dev_Q, i, chunk_size);
        int inf = hipDeviceSynchronize();
        if(inf != 0) return inf;
        printProgress((double)(i+1)/(INITIAL_IONS/chunk_size));
    }
    printf("\n");
    return 0;
}

int ion_placement(float r){
    float* hst_ini_ions_xs = (float*)malloc(INITIAL_IONS*sizeof(float));
    float* hst_ini_ions_ys = (float*)malloc(INITIAL_IONS*sizeof(float));
    float* hst_ions_placed_xs = (float*)malloc((MAX_IONS - INITIAL_IONS)*sizeof(float));
    float* hst_ions_placed_ys = (float*)malloc((MAX_IONS - INITIAL_IONS)*sizeof(float));
    populate(hst_ini_ions_xs, hst_ini_ions_ys);
    hipMemcpyToSymbol(HIP_SYMBOL(dev_ini_ions_xs), hst_ini_ions_xs, INITIAL_IONS*sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(dev_ini_ions_ys), hst_ini_ions_ys, INITIAL_IONS*sizeof(float), 0, hipMemcpyHostToDevice);
    float* dev_Q;
    float* new_ions_xs;
    float* new_ions_ys;
    int* partial_min_pos;
    float* partial_min;
    hipMalloc(&partial_min, (SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE)*sizeof(float));
    hipMalloc(&partial_min_pos, (SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE)*sizeof(int));
    hipMalloc(&dev_Q, SIZE_MALLA*SIZE_MALLA*sizeof(float));
    hipMemset(dev_Q, 0, SIZE_MALLA*SIZE_MALLA*sizeof(float));
    hipMalloc(&new_ions_xs,(MAX_IONS - INITIAL_IONS)*sizeof(float));
    hipMalloc(&new_ions_ys,(MAX_IONS - INITIAL_IONS)*sizeof(float));
    if(r == INFINITY){
        set_Qs<<<SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE, BLOCK_SIZE>>>(dev_Q);    
    }
    else{
        set_Qs_r<<<SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE, BLOCK_SIZE>>>(dev_Q, r);    
    }
    printf("Working in set_Qs\n");
    int inf = hipDeviceSynchronize();
    if(inf != 0) {printf("fail0\n"); return inf;}
    setIon(0, dev_Q, new_ions_xs, new_ions_ys, partial_min, partial_min_pos);
    printf("Updating Qs\n");
    for(int i = 1; i< MAX_IONS-INITIAL_IONS; i++){
        if(r == INFINITY){
            update_Qs<<<SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE, BLOCK_SIZE>>>(i-1, dev_Q, new_ions_xs, new_ions_ys);
        }
        else{
            update_Qs_r<<<SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE, BLOCK_SIZE>>>(i-1, dev_Q, new_ions_xs, new_ions_ys, r);
        }
        inf = hipDeviceSynchronize();
        if(inf != 0) {printf("fail1\n"); return inf;}
        setIon(i, dev_Q, new_ions_xs, new_ions_ys, partial_min, partial_min_pos);
        printProgress((double)(i+1)/(MAX_IONS-INITIAL_IONS));
    }

    inf = hipMemcpy(hst_ions_placed_xs, new_ions_xs, (MAX_IONS-INITIAL_IONS)*sizeof(float), hipMemcpyDeviceToHost);
    if(inf != 0) return inf;
    inf = hipMemcpy(hst_ions_placed_ys, new_ions_ys, (MAX_IONS-INITIAL_IONS)*sizeof(float), hipMemcpyDeviceToHost);
    if(inf != 0) return inf;

    for(int i = 0; i<MAX_IONS-INITIAL_IONS; i++){
        printf("\nION %i: (%f,%f)", i, hst_ions_placed_xs[i], hst_ions_placed_ys[i]);
    }

    hipFree(dev_Q);
    hipFree(new_ions_xs);
    hipFree(new_ions_ys);
    hipFree(partial_min);
    hipFree(partial_min_pos);
    free(hst_ini_ions_xs);
    free(hst_ini_ions_ys);
    free(hst_ions_placed_xs);
    free(hst_ions_placed_ys);
    return 0;
}

int ion_placement_only_updates(){
    float* hst_ini_ions_xs = (float*)malloc(INITIAL_IONS*sizeof(float));
    float* hst_ini_ions_ys = (float*)malloc(INITIAL_IONS*sizeof(float));
    float* hst_ions_placed_xs = (float*)malloc((MAX_IONS - INITIAL_IONS)*sizeof(float));
    float* hst_ions_placed_ys = (float*)malloc((MAX_IONS - INITIAL_IONS)*sizeof(float));
    populate(hst_ini_ions_xs, hst_ini_ions_ys);
    hipMemcpyToSymbol(HIP_SYMBOL(dev_ini_ions_xs), hst_ini_ions_xs, INITIAL_IONS*sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(dev_ini_ions_ys), hst_ini_ions_ys, INITIAL_IONS*sizeof(float), 0, hipMemcpyHostToDevice);
    float* dev_Q;
    float* new_ions_xs;
    float* new_ions_ys;
    int* partial_min_pos;
    float* partial_min;
    hipMalloc(&partial_min, (SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE)*sizeof(float));
    hipMalloc(&partial_min_pos, (SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE)*sizeof(int));
    hipMalloc(&dev_Q, SIZE_MALLA*SIZE_MALLA*sizeof(float));
    hipMemset(dev_Q, 0, SIZE_MALLA*SIZE_MALLA*sizeof(float));
    hipMalloc(&new_ions_xs,(MAX_IONS - INITIAL_IONS)*sizeof(float));
    hipMalloc(&new_ions_ys,(MAX_IONS - INITIAL_IONS)*sizeof(float));
    printf("Working in set_Qs\n");
    int inf = set_Qs_in_chunks(dev_Q);
    if(inf != 0) {printf("fail0\n"); return inf;}
    setIon(0, dev_Q, new_ions_xs, new_ions_ys, partial_min, partial_min_pos);
    printf("Updating Qs\n");
    for(int i = 1; i< MAX_IONS-INITIAL_IONS; i++){
        update_Qs<<<SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE, BLOCK_SIZE>>>(i-1, dev_Q, new_ions_xs, new_ions_ys);
        inf = hipDeviceSynchronize();
        if(inf != 0) {printf("fail1\n"); return inf;}
        setIon(i, dev_Q, new_ions_xs, new_ions_ys, partial_min, partial_min_pos);
        printProgress((double)(i+1)/(MAX_IONS-INITIAL_IONS));
    }
    inf = hipMemcpy(hst_ions_placed_xs, new_ions_xs, (MAX_IONS-INITIAL_IONS)*sizeof(float), hipMemcpyDeviceToHost);
    if(inf != 0) return inf;
    inf = hipMemcpy(hst_ions_placed_ys, new_ions_ys, (MAX_IONS-INITIAL_IONS)*sizeof(float), hipMemcpyDeviceToHost);
    if(inf != 0) return inf;
    for(int i = 0; i<MAX_IONS-INITIAL_IONS; i++){
        printf("\nION %i: (%f,%f)", i, hst_ions_placed_xs[i], hst_ions_placed_ys[i]);
    }

    hipFree(dev_Q);
    hipFree(new_ions_xs);
    hipFree(new_ions_ys);
    hipFree(partial_min);
    hipFree(partial_min_pos);
    free(hst_ini_ions_xs);
    free(hst_ini_ions_ys);
    free(hst_ions_placed_xs);
    free(hst_ions_placed_ys);
    return 0;
}

int main(){
    configSeed(1566440079);//10
    /*
    hipEvent_t ct1, ct2, ctr1, ctr2;/
    float dt1, dt2;
    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventCreate(&ctr1);
    hipEventCreate(&ctr2);
    printf("radio = INFINITY\n");
    hipEventRecord(ct1);
    int error1 = ion_placement(INFINITY);
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt1, ct1, ct2);
    //printf("\ncuda code: %i\n", error);
    //printf("Execution time all ions: %f[ms]", dt1);
    printf("\nradio = 100\n");
    hipEventRecord(ctr1);
    int error2 = ion_placement(100);
    hipEventRecord(ctr2);
    hipEventSynchronize(ctr2);
    hipEventElapsedTime(&dt2, ctr1, ctr2);
    printf("\nCUDA CODES:\n\tion_placement(INFINITY): %i \n\tion_placement(100): %i\n", error1, error2);
    printf("TIMES:\n\tion_placement(INFINITY): %f\n\tion_placement(100): %f\n", dt1, dt2);
    */
    int error = ion_placement_only_updates();
    printf("\ncuda code: %i\n", error);
    if(INFINITY == INFINITY + 3.43235) printf("YES!\n");
    return 0;
}