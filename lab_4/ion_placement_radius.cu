#include <stdio.h>
#include <stdlib.h>

#include "launch_config.cuh"
#include "reduction.cuh"
#include "kernels.cuh"

float uniform_rand(){
    return SIZE_MALLA*((float) rand() / (RAND_MAX));
}

void configSeed(unsigned seed){
    if(seed == 0){
        seed = (unsigned)time(NULL);
    }
    srand(seed);
    printf("results for seed %i\n", seed);  
}

void populate(float* ions_xs, float* ions_ys){
    for(int i = 0; i<INITIAL_IONS; i++){
        ions_xs[i]=uniform_rand();
        ions_ys[i]=uniform_rand();
    }
}

void setIon(int i, float* dev_Q, float* new_ions_xs, float* new_ions_ys, float* partial_min, int* partial_min_pos){
    int size;
    Q_reduction<<<SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE*(sizeof(float) + sizeof(int))>>>(dev_Q, partial_min, partial_min_pos);
    //printf("Ion %i\n", i);
    for(size = SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE; size>BLOCK_SIZE; size/=BLOCK_SIZE){
        //printf("size = %i\n", size);
        partial_reduction<<<size/BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE*(sizeof(float) + sizeof(int))>>>(partial_min, partial_min_pos);
        int inf = hipDeviceSynchronize();
        if(inf != 0){printf("fail in partial_reduction, cuda code: %i\n", inf);}
    }
    //printf("size = %i\n", size);
    int sMemSize = size*(sizeof(float) + sizeof(int));
    set_new_Ion<<<1, size, sMemSize>>>(i, dev_Q, new_ions_xs, new_ions_ys, partial_min, partial_min_pos);
    int inf = hipDeviceSynchronize();
    if(inf != 0){printf("fail in set_new_ion, cuda code: %i\n", inf);}
}

void printProgress (double percentage)
{
    int val = (int) (percentage * 100);
    int lpad = (int) (percentage * PBWIDTH);
    int rpad = PBWIDTH - lpad;
    printf ("\r%3d%% [%.*s%*s]", val, lpad, PBSTR, rpad, "");
    fflush (stdout);
}

void set_Qs_in_chunks(float* dev_Q, float r){
    int chunk_size = 50;
    for(int i = 0; i<INITIAL_IONS/chunk_size; i++){
        update_Qs_by_chunk_r<<<SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE, BLOCK_SIZE>>>(dev_Q, i, chunk_size, r);
        hipDeviceSynchronize();
        printProgress((double)(i+1)/(INITIAL_IONS/chunk_size));
    }
    printf("\n");
}

int ion_placement(float r){
    float* hst_ini_ions_xs = (float*)malloc(INITIAL_IONS*sizeof(float));
    float* hst_ini_ions_ys = (float*)malloc(INITIAL_IONS*sizeof(float));
    float* hst_ions_placed_xs = (float*)malloc((MAX_IONS - INITIAL_IONS)*sizeof(float));
    float* hst_ions_placed_ys = (float*)malloc((MAX_IONS - INITIAL_IONS)*sizeof(float));
    populate(hst_ini_ions_xs, hst_ini_ions_ys);
    hipMemcpyToSymbol(HIP_SYMBOL(dev_ini_ions_xs), hst_ini_ions_xs, INITIAL_IONS*sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(dev_ini_ions_ys), hst_ini_ions_ys, INITIAL_IONS*sizeof(float), 0, hipMemcpyHostToDevice);
    float* dev_Q;
    float* new_ions_xs;
    float* new_ions_ys;
    int* partial_min_pos;
    float* partial_min;
    hipMalloc(&partial_min, (SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE)*sizeof(float));
    hipMalloc(&partial_min_pos, (SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE)*sizeof(int));
    hipMalloc(&dev_Q, SIZE_MALLA*SIZE_MALLA*sizeof(float));
    hipMemset(dev_Q, 0, SIZE_MALLA*SIZE_MALLA*sizeof(float));
    hipMalloc(&new_ions_xs,(MAX_IONS - INITIAL_IONS)*sizeof(float));
    hipMalloc(&new_ions_ys,(MAX_IONS - INITIAL_IONS)*sizeof(float));
    printf("Working in set_Qs\n");
    set_Qs_in_chunks(dev_Q, r);
    setIon(0, dev_Q, new_ions_xs, new_ions_ys, partial_min, partial_min_pos);
    printf("Updating Qs\n");
    for(int i = 1; i< MAX_IONS-INITIAL_IONS; i++){
        update_Qs_r<<<SIZE_MALLA*SIZE_MALLA/BLOCK_SIZE, BLOCK_SIZE>>>(i-1, dev_Q, new_ions_xs, new_ions_ys, r);
        setIon(i, dev_Q, new_ions_xs, new_ions_ys, partial_min, partial_min_pos);
        hipDeviceSynchronize();
        printProgress((double)(i+1)/(MAX_IONS-INITIAL_IONS));
    }
    int inf = hipMemcpy(hst_ions_placed_xs, new_ions_xs, (MAX_IONS-INITIAL_IONS)*sizeof(float), hipMemcpyDeviceToHost);
    if(inf != 0) return inf;
    inf = hipMemcpy(hst_ions_placed_ys, new_ions_ys, (MAX_IONS-INITIAL_IONS)*sizeof(float), hipMemcpyDeviceToHost);
    if(inf != 0) return inf;
    for(int i = 0; i<MAX_IONS-INITIAL_IONS; i++){
        printf("\nION %i: (%f,%f)", i, hst_ions_placed_xs[i], hst_ions_placed_ys[i]);
    }
    hipFree(dev_Q);
    hipFree(new_ions_xs);
    hipFree(new_ions_ys);
    hipFree(partial_min);
    hipFree(partial_min_pos);
    free(hst_ini_ions_xs);
    free(hst_ini_ions_ys);
    free(hst_ions_placed_xs);
    free(hst_ions_placed_ys);
    return 0;
}

int main(){
    float radius = 100
    configSeed(1566440079);
    hipEvent_t ct1, ct2;
    float dt1;
    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    printf("radio = %f\n", radius);
    hipEventRecord(ct1);
    int error = ion_placement(radius);
    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt1, ct1, ct2);
    printf("\n\nCuda code: %i\n", error);
    printf("Total Time %f[ms]\n", dt1);
    return 0;
}