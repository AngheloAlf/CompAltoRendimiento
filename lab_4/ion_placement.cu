#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define GRID_SIZE 1000
#define INITIAL_IONS 5000
#define MAX_IONS 6000
#define THREADS_PER_BLOCK 512

typedef struct Ions Ions;

struct Ions{
    float xs[MAX_IONS];
    float ys[MAX_IONS];
};

float Q[GRID_SIZE*GRID_SIZE];

Ions * hst_Ions = (Ions*)malloc(sizeof(Ions));

float uniform_rand(){
    return GRID_SIZE*((float) rand() / (RAND_MAX));
}

void configSeed(){
    unsigned seed = (unsigned)time(NULL);
    srand(seed);
    printf("results for seed %i\n", seed);
}

void populateIons(){
    for(int i = 0; i<INITIAL_IONS; i++){
        hst_Ions->xs[i] = uniform_rand();
        hst_Ions->ys[i] = uniform_rand();
    }
}

void print_first_5(){
    for(int i = 0; i<5; i++){
        printf("(%f,%f)\n", hst_Ions->xs[i], hst_Ions->ys[i]);
    }
}

__device__ float distance(float p_1x, float p_1y, float p_2x, float p_2y){
    return sqrtf(powf(p_1x - p_2x, 2) + powf(p_1y - p_2y,2));
}

__global__ void update_Qs(float * dev_Q, Ions * dev_Ions, int iter){
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int x = tId%GRID_SIZE;
    int y = tId/GRID_SIZE;
    dev_Q[tId] += 1 / distance((float)x, (float)(y), dev_Ions->xs[INITIAL_IONS + iter], dev_Ions->ys[INITIAL_IONS + iter]);
}

__global__ void set_Qs(float * dev_Q, Ions * dev_Ions){
        int tId = threadIdx.x + blockIdx.x * blockDim.x;
        if(tId < GRID_SIZE*GRID_SIZE){
            int x = tId%GRID_SIZE;
            int y = tId/GRID_SIZE;
            float q = 0;
            for(int i = 0; i<INITIAL_IONS; i++){
                q += 1 / distance((float)x, (float)y, dev_Ions->xs[i], dev_Ions->ys[i]);
            }
            dev_Q[tId] = q;      
        }
}

int is_in_Ions(int limit, int pos){
    for(int i = INITIAL_IONS; i<INITIAL_IONS + limit; i++){
        if((int)hst_Ions->xs[i] + (int)hst_Ions->ys[i] * GRID_SIZE == pos){
            return 1;
        }
    }
    return 0;
}

int main(){

    float *dev_Q;
    Ions *dev_Ions;
    configSeed();
    populateIons();
    hipMalloc(&dev_Ions, sizeof(Ions));
    hipMalloc(&dev_Q, GRID_SIZE*GRID_SIZE * sizeof(float));
    for(int i=0; i<MAX_IONS-INITIAL_IONS; i++){
        hipMemcpy(dev_Ions, Ions, sizeof(Ions), hipMemcpyHostToDevice);
        if(i==0){
            set_Qs<<<GRID_SIZE*GRID_SIZE/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dev_Q, dev_Ions);
        }else{
            update_Qs<<<GRID_SIZE*GRID_SIZE/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dev_Q, dev_Ions, i);
        }
        hipMemcpy(Q, dev_Q, GRID_SIZE*GRID_SIZE * sizeof(float), hipMemcpyDeviceToHost);
        float min = INFINITY;
        for(int n = 0; n<GRID_SIZE*GRID_SIZE; n++){
            if(is_in_Ions(i,n) != 1){
                int x = n%GRID_SIZE;
                int y = n/GRID_SIZE;
                if(Q[n]<min){
                    min = Q[n];
                    hst_Ions->xs[INITIAL_IONS+i] = (float)x;
                    hst_Ions->ys[INITIAL_IONS+i] = (float)y;
                }
            }
        }
    }
    hipFree(dev_Ions);
    hipFree(dev_Q);
    return 0;
}